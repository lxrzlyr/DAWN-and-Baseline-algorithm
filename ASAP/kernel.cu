﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include "hipblas.h"
#include <time.h>
#include <iostream>
#include <fstream>
#include <omp.h>
#include "access.h"
#include "function.h"
using namespace std;


int main()
{

    int n = 0;
    int select = 1;
    cout << "请输入矩阵的阶数" << endl;
    cin >> n;
    cout << "请选择是否图的类型：1、无权；2、有权"<<endl;
    cin >> select;
   
    long long** A = new long long* [n];
    long long** B = new long long* [n];
    long long** amt = new long long* [n];
    long long** len = new long long* [n];
    for (int i = 0; i < n; i++)
    {
        A[i] = new long long[n];
        B[i] = new long long[n];
        amt[i] = new long long[n];
        len[i] = new long long[n];
        for (int j = 0; j < n; j++)
        {
            A[i][j] = 0;
            B[i][j] = 0;
            amt[i][j] = 0;
            len[i][j] = 0;
        }
    }
    if (select == 1)
    {
        ifstream fin("matrix.txt");
        for (int i = 0; i < n; i++)
            for (int j = 0; j < n; j++)
            {
                fin >> A[i][j];
                if (A[i][j] == 1)
                {
                    B[i][j] =1;
                    amt[i][j] =1;
                    len[i][j] = 1;
                }
                else
                {
                    B[i][j] = 0;
                    amt[i][j] = -1;
                    len[i][j] = -1;
                }
            }
        cout << "程序已读入数据" << endl;
        unweighted(A, B, amt, len, n);
    }
    if (select == 2)
    {
        ifstream fin("matrix.txt");
        map<pair<int, int>, int>mp;
        for (int i = 0; i < n; i++)
            for (int j = 0; j < n; j++)
            {
                fin >> A[i][j];

                if (A[i][j] == 1)
                {
                    B[i][j] = 1;
                    amt[i][j] = 1;
                    len[i][j] = 1;
                }
                else
                {
                    B[i][j] = 0;
                    amt[i][j] = -1;
                    len[i][j] = -1;
                    if (A[i][j] > 1)
                        mp[{i, j}] = A[i][j];
                }
            }
        cout << "程序已读入数据" << endl;
        weighted(A, B, amt, len, n,  mp);
    }
    ofstream fout2("length.txt");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
            fout2 << len[i][j] << " ";
        fout2 << endl;
    }
    cout << "len数据打印完毕" << endl;
    ofstream fout("amount.txt");
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
            fout << amt[i][j] << " ";
        fout << endl;
    }
    

    for (int i = 0; i < n; i++)
    {
        delete[]A[i];
        delete[]B[i];
        delete[]amt[i];
    }
    delete[]A;
    delete[]B;
    delete[]amt;



    return 0;
}

